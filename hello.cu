/*
 * Hello World in CUDA
 *
 * CS3210
 *
 * This program should print "HELLO WORLD" if successful.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N       32 

// #define      DISCRETE

__global__ void hello(char *a, int len)
{       int threadId = threadIdx.x * blockDim.y + threadIdx.y; 
        int blockId = blockIdx.x * gridDim.y * gridDim.z + blockIdx.y * gridDim.z + blockIdx.z;
        int tid = (blockId) * blockDim.x * blockDim.y + threadId;
        if (tid >= len)
                return;
        a[tid] += 'A' - 'a';
}
// device: GPU 
// host: CPU 

int main()
{
        // original string
        char a[N] = "hello@world";
        // length
        int len = strlen(a);
        // pointer to the string on device
        char* ad;
        // pointer to the final string on host
        char* ah;
        // CUDA returned error code
        hipError_t rc;


        //allocate space for the string on device (GPU) memory
        hipMalloc((void**)&ad, N);
        hipMemcpy(ad, a, N, hipMemcpyHostToDevice);
        dim3 gridDimensions(2, 2, 2); 
        dim3 blockDimensions(2, 4);
        // launch the kernel
        hello<<<gridDimensions, blockDimensions>>>(ad, len);
        hipDeviceSynchronize();

	// for discrete GPUs, get the data from device memory to host memory
        hipMemcpy(a, ad, N, hipMemcpyDeviceToHost);
        ah = a;

        // was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

        // print final string
        printf("%s!\n", ah);

        // free memory
        hipFree(ad);

        return 0;
}

